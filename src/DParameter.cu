#include "hip/hip_runtime.h"
#include "DParameter.cuh"
#include "ChemData.h"
#include <filesystem>
#include "Constants.h"

cfd::DParameter::DParameter(Parameter &parameter, const Species &species, Reaction *reaction) :
  myid{parameter.get_int("myid")},
  dim{parameter.get_int("dimension")}, problem_type{parameter.get_int("problem_type")},
  n_var{parameter.get_int("n_var")}, n_scalar_transported{parameter.get_int("n_scalar_transported")},
  i_fl{parameter.get_int("i_fl")}, i_fl_cv{parameter.get_int("i_fl_cv")},
  i_turb_cv{parameter.get_int("i_turb_cv")}, inviscid_scheme{parameter.get_int("inviscid_scheme")},
  reconstruction{parameter.get_int("reconstruction")}, limiter{parameter.get_int("limiter")},
  viscous_scheme{parameter.get_int("viscous_order")}, positive_preserving{parameter.get_bool("positive_preserving")},
  gradPInDiffusionFlux{parameter.get_bool("gradPInDiffusionFlux")},
  entropy_fix_factor{parameter.get_real("entropy_fix_factor")}, fixed_dt{parameter.get_bool("fixed_time_step")},
  rans_model{parameter.get_int("RANS_model")}, turb_implicit{parameter.get_int("turb_implicit")},
  compressibility_correction{parameter.get_int("compressibility_correction")},
  chemSrcMethod{parameter.get_int("chemSrcMethod")},
  Pr(parameter.get_real("prandtl_number")), cfl(parameter.get_real("cfl")),
  Prt(parameter.get_real("turbulent_prandtl_number")),
  Sct(parameter.get_real("turbulent_schmidt_number")),
  rho_ref{parameter.get_real("rho_inf")},
  a_ref2{parameter.get_real("speed_of_sound") * parameter.get_real("speed_of_sound")},
  v_ref{parameter.get_real("v_inf")}, T_ref{parameter.get_real("T_inf")}, mach_ref{parameter.get_real("M_inf")},
  p_ref{parameter.get_real("p_inf")}, weno_eps_scale{
    parameter.get_real("rho_inf") * parameter.get_real("v_inf") * parameter.get_real("rho_inf") *
    parameter.get_real("v_inf")
  }, perform_spanwise_average{parameter.get_bool("perform_spanwise_average")}
/*,sponge_layer{parameter.get_bool("sponge_layer")}, sponge_function{parameter.get_int("sponge_function")},
sponge_iter{parameter.get_int("sponge_iter")}, spongeXMinusStart{parameter.get_real("spongeXMinusStart")},
spongeXMinusEnd{parameter.get_real("spongeXMinusEnd")}, spongeXPlusStart{parameter.get_real("spongeXPlusStart")},
spongeXPlusEnd{parameter.get_real("spongeXPlusEnd")}, spongeYMinusStart{parameter.get_real("spongeYMinusStart")},
spongeYMinusEnd{parameter.get_real("spongeYMinusEnd")}, spongeYPlusStart{parameter.get_real("spongeYPlusStart")},
spongeYPlusEnd{parameter.get_real("spongeYPlusEnd")}, spongeZMinusStart{parameter.get_real("spongeZMinusStart")},
spongeZMinusEnd{parameter.get_real("spongeZMinusEnd")}, spongeZPlusStart{parameter.get_real("spongeZPlusStart")},
spongeZPlusEnd{parameter.get_real("spongeZPlusEnd")}*/ {
  if (parameter.get_int("myid") == 0) {
    if (inviscid_scheme == 51 || inviscid_scheme == 52 || inviscid_scheme == 71 || inviscid_scheme == 72)
      printf("\t->-> %-20e : WENO scale factor\n", weno_eps_scale);
  }

  if (parameter.get_bool("fixed_time_step")) {
    dt = parameter.get_real("dt");
  }

  // if (parameter.get_string("hybrid_inviscid_scheme") != "NO") {
    shock_sensor = parameter.get_int("shock_sensor");
    sensor_eps = parameter.get_real("shockSensor_epsilon");
    sensor_threshold = parameter.get_real("shockSensor_threshold");
  // }

  const auto &spec = species;
  n_spec = spec.n_spec;
  n_scalar = parameter.get_int("n_scalar");
  if (reaction != nullptr) {
    n_reac = reaction->n_reac;
  }
  n_ps = parameter.get_int("n_ps");
  i_ps = parameter.get_int("i_ps");
  i_ps_cv = parameter.get_int("i_ps_cv");
  if (n_ps > 0) {
    hipMalloc(&sc_ps, n_ps * sizeof(real));
    hipMemcpy(sc_ps, parameter.get_real_array("sc_passive_scalar").data(), n_ps * sizeof(real),
               hipMemcpyHostToDevice);
    hipMalloc(&sct_ps, n_ps * sizeof(real));
    hipMemcpy(sct_ps, parameter.get_real_array("sct_passive_scalar").data(), n_ps * sizeof(real),
               hipMemcpyHostToDevice);
  }

  // species info
  auto mem_sz = n_spec * sizeof(real);
  // hipMalloc(&mw, mem_sz);
  // hipMemcpy(mw, spec.mw.data(), mem_sz, hipMemcpyHostToDevice);
  std::vector<real> imw_vec(n_spec), Rl(n_spec);
  for (int l = 0; l < n_spec; ++l) {
    imw_vec[l] = 1.0 / spec.mw[l];
    Rl[l] = R_u / spec.mw[l];
  }
  hipMalloc(&imw, mem_sz);
  hipMemcpy(imw, imw_vec.data(), mem_sz, hipMemcpyHostToDevice);
  hipMalloc(&gas_const, mem_sz);
  hipMemcpy(gas_const, Rl.data(), mem_sz, hipMemcpyHostToDevice);
  #ifdef HighTempMultiPart
  hipMalloc(&n_temperature_range, n_spec * sizeof(int));
  hipMemcpy(n_temperature_range, spec.n_temperature_range.data(), n_spec * sizeof(int), hipMemcpyHostToDevice);
  int n_ranges = 2;
  for (int l = 0; l < n_spec; ++l) {
    n_ranges = std::max(spec.n_temperature_range[l], n_ranges);
  }
  temperature_cuts.init_with_size(n_spec, n_ranges + 1);
  hipMemcpy(temperature_cuts.data(), spec.temperature_range.data(), temperature_cuts.size() * sizeof(real),
             hipMemcpyHostToDevice);
  therm_poly_coeff.allocate_memory(7, n_ranges, n_spec, 0);
  hipMemcpy(therm_poly_coeff.data(), spec.therm_poly_coeff.data(), sizeof(real) * therm_poly_coeff.size(),
             hipMemcpyHostToDevice);
  #else
  high_temp_coeff.init_with_size(n_spec, 7);
  hipMemcpy(high_temp_coeff.data(), spec.high_temp_coeff.data(), high_temp_coeff.size() * sizeof(real),
             hipMemcpyHostToDevice);
  low_temp_coeff.init_with_size(n_spec, 7);
  hipMemcpy(low_temp_coeff.data(), spec.low_temp_coeff.data(), low_temp_coeff.size() * sizeof(real),
             hipMemcpyHostToDevice);
  hipMalloc(&t_low, mem_sz);
  hipMalloc(&t_mid, mem_sz);
  hipMalloc(&t_high, mem_sz);
  hipMemcpy(t_low, spec.t_low.data(), mem_sz, hipMemcpyHostToDevice);
  hipMemcpy(t_mid, spec.t_mid.data(), mem_sz, hipMemcpyHostToDevice);
  hipMemcpy(t_high, spec.t_high.data(), mem_sz, hipMemcpyHostToDevice);
  #endif
  hipMalloc(&geometry, mem_sz);
  hipMemcpy(geometry, spec.geometry.data(), mem_sz, hipMemcpyHostToDevice);
  hipMalloc(&LJ_potent_inv, mem_sz);
  hipMemcpy(LJ_potent_inv, spec.LJ_potent_inv.data(), mem_sz, hipMemcpyHostToDevice);
  hipMalloc(&vis_coeff, mem_sz);
  hipMemcpy(vis_coeff, spec.vis_coeff.data(), mem_sz, hipMemcpyHostToDevice);
  WjDivWi_to_One4th.init_with_size(n_spec, n_spec);
  hipMemcpy(WjDivWi_to_One4th.data(), spec.WjDivWi_to_One4th.data(), WjDivWi_to_One4th.size() * sizeof(real),
             hipMemcpyHostToDevice);
  sqrt_WiDivWjPl1Mul8.init_with_size(n_spec, n_spec);
  hipMemcpy(sqrt_WiDivWjPl1Mul8.data(), spec.sqrt_WiDivWjPl1Mul8.data(),
             sqrt_WiDivWjPl1Mul8.size() * sizeof(real), hipMemcpyHostToDevice);
  binary_diffusivity_coeff.init_with_size(n_spec, n_spec);
  hipMemcpy(binary_diffusivity_coeff.data(), spec.binary_diffusivity_coeff.data(),
             binary_diffusivity_coeff.size() * sizeof(real), hipMemcpyHostToDevice);
  kb_over_eps_jk.init_with_size(n_spec, n_spec);
  hipMemcpy(kb_over_eps_jk.data(), spec.kb_over_eps_jk.data(),
             kb_over_eps_jk.size() * sizeof(real), hipMemcpyHostToDevice);
  hipMalloc(&ZRotF298, mem_sz);
  hipMemcpy(ZRotF298, spec.ZRotF298.data(), mem_sz, hipMemcpyHostToDevice);
  Sc = parameter.get_real("schmidt_number");

  // reaction info
  if (n_reac > 0) {
    hipMalloc(&reac_type, n_reac * sizeof(int));
    hipMemcpy(reac_type, reaction->label.data(), n_reac * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc(&rev_type, n_reac * sizeof(int));
    hipMemcpy(rev_type, reaction->rev_type.data(), n_reac * sizeof(int), hipMemcpyHostToDevice);
    stoi_f.init_with_size(n_reac, n_spec);
    hipMemcpy(stoi_f.data(), reaction->stoi_f.data(), stoi_f.size() * sizeof(int), hipMemcpyHostToDevice);
    stoi_b.init_with_size(n_reac, n_spec);
    hipMemcpy(stoi_b.data(), reaction->stoi_b.data(), stoi_b.size() * sizeof(int), hipMemcpyHostToDevice);
    mem_sz = n_reac * sizeof(real);
    hipMalloc(&reac_order, n_reac * sizeof(int));
    hipMemcpy(reac_order, reaction->order.data(), n_reac * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc(&A, mem_sz);
    hipMemcpy(A, reaction->A.data(), mem_sz, hipMemcpyHostToDevice);
    hipMalloc(&b, mem_sz);
    hipMemcpy(b, reaction->b.data(), mem_sz, hipMemcpyHostToDevice);
    hipMalloc(&Ea, mem_sz);
    hipMemcpy(Ea, reaction->Ea.data(), mem_sz, hipMemcpyHostToDevice);
    hipMalloc(&A2, mem_sz);
    hipMemcpy(A2, reaction->A2.data(), mem_sz, hipMemcpyHostToDevice);
    hipMalloc(&b2, mem_sz);
    hipMemcpy(b2, reaction->b2.data(), mem_sz, hipMemcpyHostToDevice);
    hipMalloc(&Ea2, mem_sz);
    hipMemcpy(Ea2, reaction->Ea2.data(), mem_sz, hipMemcpyHostToDevice);
    third_body_coeff.init_with_size(n_reac, n_spec);
    hipMemcpy(third_body_coeff.data(), reaction->third_body_coeff.data(), third_body_coeff.size() * sizeof(real),
               hipMemcpyHostToDevice);
    hipMalloc(&troe_alpha, mem_sz);
    hipMemcpy(troe_alpha, reaction->troe_alpha.data(), mem_sz, hipMemcpyHostToDevice);
    hipMalloc(&troe_t3, mem_sz);
    hipMemcpy(troe_t3, reaction->troe_t3.data(), mem_sz, hipMemcpyHostToDevice);
    hipMalloc(&troe_t1, mem_sz);
    hipMemcpy(troe_t1, reaction->troe_t1.data(), mem_sz, hipMemcpyHostToDevice);
    hipMalloc(&troe_t2, mem_sz);
    hipMemcpy(troe_t2, reaction->troe_t2.data(), mem_sz, hipMemcpyHostToDevice);
  }

  if (parameter.get_bool("if_collect_statistics")) {
    n_reyAve = static_cast<int>(parameter.get_int_array("reyAveVarIndex").size());
    hipMalloc(&reyAveVarIndex, n_reyAve * sizeof(int));
    hipMemcpy(reyAveVarIndex, parameter.get_int_array("reyAveVarIndex").data(), n_reyAve * sizeof(int),
               hipMemcpyHostToDevice);
    n_reyAveScalar = static_cast<int>(parameter.get_int_array("reyAveScalarIndex").size());
    hipMalloc(&reyAveScalarIndex, n_reyAveScalar * sizeof(int));
    hipMemcpy(reyAveScalarIndex, parameter.get_int_array("reyAveScalarIndex").data(), n_reyAveScalar * sizeof(int),
               hipMemcpyHostToDevice);
    rho_p_correlation = parameter.get_bool("rho_p_correlation");
    stat_tke_budget = parameter.get_bool("stat_tke_budget");
    stat_scalar_fluc_budget = parameter.get_bool("stat_scalar_fluc_budget");
    stat_species_dissipation_rate = parameter.get_bool("stat_species_dissipation_rate");
    stat_species_velocity_correlation = parameter.get_bool("stat_species_velocity_correlation");
    n_species_stat = parameter.get_int("n_species_stat");
    hipMalloc(&specStatIndex, n_species_stat * sizeof(int));
    hipMemcpy(specStatIndex, parameter.get_int_array("species_stat_index").data(), n_species_stat * sizeof(int),
               hipMemcpyHostToDevice);
  }

  if (parameter.get_string("canonical_problem") == "jicf") {
    n_jet = parameter.get_int("n_jet");
    jet_radius = parameter.get_real("jet_radius");
    if (n_jet > 0) {
      hipMalloc(&xc_jet, n_jet * sizeof(real));
      hipMemcpy(xc_jet, parameter.get_real_array("xc_jet").data(), n_jet * sizeof(real), hipMemcpyHostToDevice);
      hipMalloc(&zc_jet, n_jet * sizeof(real));
      hipMemcpy(zc_jet, parameter.get_real_array("zc_jet").data(), n_jet * sizeof(real), hipMemcpyHostToDevice);

      // jet info
      std::vector<real> jet_uh(n_jet), jet_vh(n_jet), jet_wh(n_jet), jet_Th(n_jet), jet_ph(n_jet), jet_rhoh(n_jet);
      gxl::MatrixDyn<real> jet_svh;
      jet_svh.resize(n_jet, n_spec);
      for (int i = 0; i < n_jet; i++) {
        auto struct_name = "jet-" + std::to_string(i);
        auto info = parameter.get_struct(struct_name);
        auto jet1_mach = std::get<real>(info.at("mach"));
        auto jet1_u = std::get<real>(info.at("u"));
        auto jet1_v = std::get<real>(info.at("v"));
        auto jet1_w = std::get<real>(info.at("w"));
        auto jet1_T = std::get<real>(info.at("temperature"));
        auto jet1_p = std::get<real>(info.at("pressure"));
        real jet1_rho;
        auto jet1_sv = new real[n_spec];
        memset(jet1_sv, 0, n_spec * sizeof(real));
        if (n_spec > 0) {
          // const auto& spec_list = spec.spec_list;
          real mw_inv = 0;
          for (const auto &[name, idx]: spec.spec_list) {
            if (info.find(name) != info.cend()) {
              jet1_sv[idx] = std::get<real>(info.at(name));
              // printf("%s = %e\n", name.c_str(), jet1_sv[idx]);
            }
          }
          for (int l = 0; l < n_spec; ++l) {
            mw_inv += jet1_sv[l] / spec.mw[l];
          }
          jet1_rho = jet1_p / (R_u * mw_inv * jet1_T);
          // printf("rho = %e\n", jet1_rho);
          std::vector<real> cpi(n_spec, 0);
          spec.compute_cp(jet1_T, cpi.data());
          real cp{0}, cv{0};
          for (int l = 0; l < n_spec; ++l) {
            cp += jet1_sv[l] * cpi[l];
            cv += jet1_sv[l] * (cpi[l] - R_u / spec.mw[l]);
          }
          real gamma = cp / cv; // specific heat ratio
          real c = sqrt(gamma * R_u * jet1_T * mw_inv);
          // printf("cp = %f, cv = %f, gamma = %f, c=%f\n", cp, cv, gamma, c);
          jet1_u *= jet1_mach * c;
          jet1_v *= jet1_mach * c;
          jet1_w *= jet1_mach * c;
        } else {
          jet1_rho = jet1_p / (R_air * jet1_T);
          real c = sqrt(gamma_air * R_air * jet1_T);
          jet1_u *= jet1_mach * c;
          jet1_v *= jet1_mach * c;
          jet1_w *= jet1_mach * c;
        }
        jet_uh[i] = jet1_u;
        jet_vh[i] = jet1_v;
        jet_wh[i] = jet1_w;
        jet_Th[i] = jet1_T;
        jet_ph[i] = jet1_p;
        jet_rhoh[i] = jet1_rho;
        for (int l = 0; l < n_spec; ++l) {
          jet_svh(i, l) = jet1_sv[l];
        }
        // printf("rho=%e, u=%e, v=%e, w=%e, T=%e, p=%e\n", jet1_rho, jet1_u, jet1_v, jet1_w, jet1_T, jet1_p);
      }
      hipMalloc(&jet_u, n_jet * sizeof(real));
      hipMemcpy(jet_u, jet_uh.data(), n_jet * sizeof(real), hipMemcpyHostToDevice);
      hipMalloc(&jet_v, n_jet * sizeof(real));
      hipMemcpy(jet_v, jet_vh.data(), n_jet * sizeof(real), hipMemcpyHostToDevice);
      hipMalloc(&jet_w, n_jet * sizeof(real));
      hipMemcpy(jet_w, jet_wh.data(), n_jet * sizeof(real), hipMemcpyHostToDevice);
      hipMalloc(&jet_T, n_jet * sizeof(real));
      hipMemcpy(jet_T, jet_Th.data(), n_jet * sizeof(real), hipMemcpyHostToDevice);
      hipMalloc(&jet_p, n_jet * sizeof(real));
      hipMemcpy(jet_p, jet_ph.data(), n_jet * sizeof(real), hipMemcpyHostToDevice);
      hipMalloc(&jet_rho, n_jet * sizeof(real));
      hipMemcpy(jet_rho, jet_rhoh.data(), n_jet * sizeof(real), hipMemcpyHostToDevice);
      jet_sv.init_with_size(n_jet, n_spec);
      hipMemcpy(jet_sv.data(), jet_svh.data(), n_jet * n_spec * sizeof(real), hipMemcpyHostToDevice);
    }
  }

  // If mixing layer and multi-component, we need the mixture fraction info.
  if (problem_type == 1 && n_spec > 0) {
    beta_diff_inv = parameter.get_real("beta_diff_inv");
    beta_o = parameter.get_real("beta_o");
    nuc_mwc_inv = parameter.get_real("nuc_mwc_inv");
    nuh_mwh_inv = parameter.get_real("nuh_mwh_inv");
    half_nuo_mwo_inv = parameter.get_real("half_nuo_mwo_inv");
  }

  // the following parameters have been computed in "write_reference_state".
  if (problem_type == 1) {
    convective_velocity = parameter.get_real("convective_velocity");
    delta_omega0 = parameter.get_real("delta_omega");
    if (int i = parameter.get_int("characteristic_velocity_ml"); i == 0) {
      v_char = parameter.get_real("convective_velocity");
    } else {
      v_char = parameter.get_real("DeltaU");
    }
    delta_u = parameter.get_real("DeltaU");
  } else {
    v_char = parameter.get_real("v_inf");
  }

  fluctuation_form = parameter.get_int("fluctuation_form");
  fluctuation_intensity = parameter.get_real("fluctuation_intensity");
  N_spanwise_wave = parameter.get_int("N_spanwise_waves");
  x0_fluc = parameter.get_real("x0_fluc");
  y0_fluc = parameter.get_real("y0_fluc");
  z0_fluc = parameter.get_real("z0_fluc");

  memset(limit_flow.ll, 0, sizeof(real) * LimitFlow::max_n_var);
  memset(limit_flow.ul, 0, sizeof(real) * LimitFlow::max_n_var);
  memset(limit_flow.sv_inf, 0, sizeof(real) * (MAX_SPEC_NUMBER + 2));
  // density limits
  limit_flow.ll[0] = 1e-6 * parameter.get_real("rho_inf");
  limit_flow.ul[0] = 1e+3 * parameter.get_real("rho_inf");
  for (int l = 1; l < 4; ++l) {
    real vRef{parameter.get_real("v_inf")};
    if (abs(vRef) < 1) {
      vRef = parameter.get_real("speed_of_sound");
    }
    limit_flow.ll[l] = -1000 * vRef;
    limit_flow.ul[l] = 1000 * vRef;
  }
  // pressure limits
  limit_flow.ll[4] = 1e-6 * parameter.get_real("p_inf");
  limit_flow.ul[4] = 1e+3 * parameter.get_real("p_inf");
  if (rans_model == 2) {
    // SST model
    limit_flow.ul[5] = std::numeric_limits<real>::max();
    limit_flow.ul[6] = std::numeric_limits<real>::max();
  }
  auto &sv_inf{parameter.get_real_array("sv_inf")};
  for (int l = 0; l < n_scalar; ++l) {
    limit_flow.sv_inf[l] = sv_inf[l];
  }

  // if (parameter.get_bool("sponge_layer")) {
  //   spongeX = parameter.get_int("spongeX");
  //   spongeY = parameter.get_int("spongeY");
  //   spongeZ = parameter.get_int("spongeZ");
  //   if (parameter.get_int("n_scalar") > 0) {
  //     hipMalloc(&sponge_scalar_iter, n_scalar * sizeof(int));
  //     hipMemcpy(sponge_scalar_iter, parameter.get_int_array("sponge_scalar_iter").data(), n_scalar * sizeof(int),
  //                hipMemcpyHostToDevice);
  //   }
  //   if (spongeX == 1 || spongeX == 3) {
  //     sponge_sigma0 = parameter.get_real("spongeCoefficient") * v_char / (spongeXMinusStart - spongeXMinusEnd);
  //     printf("sponge_sigma0=%e\n", sponge_sigma0);
  //   }
  //   if (spongeX == 2 || spongeX == 3) {
  //     sponge_sigma1 = parameter.get_real("spongeCoefficient") * v_char / (spongeXPlusEnd - spongeXPlusStart);
  //     printf("sponge_sigma1=%e\n", sponge_sigma1);
  //   }
  //   if (spongeY == 1 || spongeY == 3) {
  //     sponge_sigma2 = parameter.get_real("spongeCoefficient") * v_char / (spongeYMinusStart - spongeYMinusEnd);
  //     printf("sponge_sigma2=%e\n", sponge_sigma2);
  //   }
  //   if (spongeY == 2 || spongeY == 3) {
  //     sponge_sigma3 = parameter.get_real("spongeCoefficient") * v_char / (spongeYPlusEnd - spongeYPlusStart);
  //     printf("sponge_sigma3=%e\n", sponge_sigma3);
  //   }
  //   if (spongeZ == 1 || spongeZ == 3) {
  //     sponge_sigma4 = parameter.get_real("spongeCoefficient") * v_char / (spongeZMinusStart - spongeZMinusEnd);
  //     printf("sponge_sigma4=%e\n", sponge_sigma4);
  //   }
  //   if (spongeZ == 2 || spongeZ == 3) {
  //     sponge_sigma5 = parameter.get_real("spongeCoefficient") * v_char / (spongeZPlusEnd - spongeZPlusStart);
  //     printf("sponge_sigma5=%e\n", sponge_sigma5);
  //   }
  // }
}

//cfd::DParameter::~DParameter() {
//  hipFree(mw);
//  hipFree(t_low);
//  hipFree(t_mid);
//  hipFree(t_high);
//  hipFree(LJ_potent_inv);
//  hipFree(vis_coeff);
//  hipFree(reac_type);
//  hipFree(reac_order);
//  hipFree(A);
//  hipFree(b);
//  hipFree(Ea);
//  hipFree(A2);
//  hipFree(b2);
//  hipFree(Ea2);
//  hipFree(troe_alpha);
//  hipFree(troe_t3);
//  hipFree(troe_t1);
//  hipFree(troe_t2);
//  hipFree(mix_frac);
//}
__global__ void cfd::update_dt_global(DParameter *param, real dt) {
  param->dt = dt;
}
